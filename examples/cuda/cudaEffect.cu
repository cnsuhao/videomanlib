#include <hip/hip_runtime.h>
#include <iostream>

// The CUDA code is not optimized

using namespace std;

unsigned char *srcd = 0;
unsigned char *dstd = 0;
unsigned char *tmpd = 0;

__device__ static void calculateRanges( int indexX, int indexY, int size, int width, int height, int &numPixels, int &beginX, int &endX, int &beginY, int &endY )
{
	beginX = indexX - size;
	if ( beginX < 0 )
		beginX = 0;
	endX = indexX + size;
	if ( endX >= width )
		endX = width - 1;
	beginY = indexY - size;
	if ( beginY < 0 )
		beginY = 0;
	endY = indexY + size;
	if ( endY >= height )
		endY = height - 1;
	numPixels = ( endX - beginX + 1 ) * ( endY - beginY + 1 );
}

__global__ void pixelizationKernel( unsigned char *src, unsigned char *dst, int width, int height, int BPP )
{
	const int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int index_y = blockIdx.y * blockDim.y + threadIdx.y;

	const int pixIndex = index_y * width + index_x;

	__shared__ int pixelVal[3];
	if ( threadIdx.x == 0 && threadIdx.y == 0 )
	{		
		for ( int c = 0; c < 3; ++c )
			pixelVal[c] = src[pixIndex * BPP + c];
	}

	__syncthreads();
	
	for ( int c = 0; c < 3; ++c )
		dst[pixIndex*BPP+c] = pixelVal[c];//src[pixIndex0*BPP+c];
}

__global__ void blurKernel( unsigned char *src, unsigned char *dst, int width, int height, int BPP )
{
	const int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int index_y = blockIdx.y * blockDim.y + threadIdx.y;

	const int pixIndex = index_y * width + index_x;
	
	int beginX, endX, beginY, endY, numPixels;
	calculateRanges( index_x, index_y, 3, width, height, numPixels, beginX, endX, beginY, endY );

	int sum[] = {0,0,0};
	for ( int f = beginY; f <= endY; ++f )
	{
		for ( int c = beginX; c <= endX; ++c )
		{
			const int index = ( f * width + c ) * BPP;
			sum[0] += src[index];
			sum[1] += src[index + 1];
			sum[2] += src[index + 2];			
		}
	}
	for ( int c = 0; c < 3; ++c )
		dst[pixIndex*BPP+c] = sum[c] / numPixels;
}

__global__ void dilationKernel( unsigned char *src, unsigned char *dst, int width, int height, int BPP )
{
	const int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	const int pixIndex = index_y * width + index_x;

	int max = 0;//[] = {0,0,0};
	int fMax = index_y - 2;
	int cMax = index_x - 1;
	int beginX, endX, beginY, endY, numPixels;
	calculateRanges( index_x, index_y, 2, width, height, numPixels, beginX, endX, beginY, endY );
	for ( int f = beginY; f < endY; ++f )
	{
		for ( int c = beginX; c < endX; ++c )
		{
			const int index = ( f * width + c ) * BPP;	
			for ( int channel = 0; channel < 3; ++channel )
			{
				const int aux = src[index + channel];
				if( aux > max )
				{
					max = aux;
					cMax = c;					
					fMax = f;
				}
			}			
		}
	}
	const int index = ( fMax * width + cMax ) * BPP;
	for ( int channel = 0; channel < 3; ++channel )	
		dst[pixIndex*BPP+channel] = src[index + channel];
}

__global__ void blocksKernel( unsigned char *src, unsigned char *dst, int width, int height, int BPP )
{
	const int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int index_y = blockIdx.y * blockDim.y + threadIdx.y;

	const int pixIndex = index_y * width + index_x;
	
	dst[pixIndex*BPP+2] = 255 * blockIdx.x / gridDim.x;//255 * threadIdx.x / blockDim.x;
	dst[pixIndex*BPP+1] = src[pixIndex*BPP+2] ;//255 * threadIdx.x / blockDim.x;
	dst[pixIndex*BPP] = 255 * blockIdx.y / gridDim.y;//255 * threadIdx.y / blockDim.y;
}

__global__ void ghostKernel( unsigned char *src, unsigned char *dst, int width, int height, int BPP )
{
	const int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int index_y = blockIdx.y * blockDim.y + threadIdx.y;

	const int pixIndex = index_y * width + index_x;
	
	//ghost effect
	for ( int c = 0; c < 3; ++c )
	{
		const float aux = src[pixIndex*BPP+c];
		float sum = ( 0.25f * aux + 0.75f * (float)dst[pixIndex*BPP+c] );
		if ( sum > 255 ) sum = 255;
		src[pixIndex*BPP+c] = sum;
		dst[pixIndex*BPP+c] = sum;
	}
}

__global__ void screeenBlendKernel( unsigned char *src, unsigned char *blend, unsigned char *dst, int width, int height, int BPP )
{
	const int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int index_y = blockIdx.y * blockDim.y + threadIdx.y;

	const int pixIndex = ( index_y * width + index_x ) * BPP;
	
	for ( int c = 0; c < 3; ++c )
		dst[pixIndex + c] = 255 - ( ( ( 255 - blend[pixIndex + c] ) * ( 255 - src[pixIndex + c] ) ) ) / 255;
}

__global__ void aberrationKernel( unsigned char *src, unsigned char *dst, int width, int height, int BPP )
{
	const int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int index_y = blockIdx.y * blockDim.y + threadIdx.y;

	//const int pixIndex = index_y * width + index_x;	
	dst[( index_y * width + index_x ) * BPP + 0] = src[(( index_y - 5 )* width + index_x -3)* BPP];
	dst[( index_y * width + index_x ) * BPP + 1] = src[(( index_y + 5 )* width + index_x +3)* BPP + 1];
	dst[( index_y * width + index_x ) * BPP + 2] = src[( index_y * width + index_x - 7)* BPP + 2];
	
}

extern "C"
void allocGPUMem( int width, int height, int BPP )
{
	const int sizeBytes = sizeof(unsigned char) * width * height * BPP;
	hipError_t err = hipMalloc( (void**)&srcd, sizeBytes );
	err = hipMalloc( (void**)&dstd, sizeBytes );	
	err = hipMalloc( (void**)&tmpd, sizeBytes );		
	err = hipMemset( srcd, 0, sizeBytes );
	err = hipMemset( dstd, 0, sizeBytes );
	err = hipMemset( tmpd, 0, sizeBytes );	
	if(err != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(err));
		exit(-1);
	}        
}

extern "C"
void freeGPUMem()
{
	hipError_t err = hipFree( srcd );
	err = hipFree( dstd );
	err = hipFree( tmpd );	
	if(err != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(err));
		exit(-1);
	}        
}

extern "C"
void executeCudaKernel( int effect, unsigned char *img, int width, int height, int BPP )
{
	const int sizeBytes = sizeof(unsigned char) * width * height * BPP;
	hipError_t err = hipMemcpy( srcd, img, sizeBytes, hipMemcpyHostToDevice );
	const int threads = 16;
	dim3 dimGrid( width / (float)threads, height / (float)threads );	
	dim3 dimBlock( threads,  threads );
	switch(effect)
	{
		case 0:
		{	
			pixelizationKernel<<< dimGrid, dimBlock >>>( srcd, dstd, width, height, BPP );
			break;
		}		
		case 1:
		{	
			blocksKernel<<< dimGrid, dimBlock >>>( srcd, dstd, width, height, BPP );
			break;
		}
		case 2:
		{	
			ghostKernel<<< dimGrid, dimBlock >>>( srcd, dstd, width, height, BPP );
			break;
		}
		case 3:
		{	
			dilationKernel<<< dimGrid, dimBlock >>>( srcd, dstd, width, height, BPP );
			break;
		}
		case 4:
		{	
			dilationKernel<<< dimGrid, dimBlock >>>( srcd, tmpd, width, height, BPP );
			blurKernel<<< dimGrid, dimBlock >>>( tmpd, dstd, width, height, BPP );
			break;
		}
		case 5:
		{
			dilationKernel<<< dimGrid, dimBlock >>>( srcd, dstd, width, height, BPP );
			blurKernel<<< dimGrid, dimBlock >>>( dstd, tmpd, width, height, BPP );
			screeenBlendKernel<<< dimGrid, dimBlock >>>( srcd, tmpd, dstd, width, height, BPP );
			break;
		}		
	}
	hipMemcpy( img, dstd, sizeBytes, hipMemcpyDeviceToHost );
	err = hipGetLastError();
	if(err != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(err));
		exit(-1);
	}                                     
}
